    
#include <hip/hip_runtime.h>
#include <iostream>
    #include <stdio.h>
    #include <math.h>


    // CUDA kernel to add elements
    __global__    void add(int N, float *x)
    {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
   
      if (i<N) 
          x[i] = x[i] *2;
    }

    int main(void)
    {
      int N = 1<<20;
      float *x;

      // Allocate Unified Memory -- accessible from CPU or GPU
      hipMallocManaged(&x, N*sizeof(float));

      // initialize x array on the host
      for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
      }

      // Launch kernel on 1M elements on the GPU
      int threadsPerBlock = 256;
      int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

      hipSetDevice(0);
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, 0);

      if( (unsigned long long) (N*sizeof(float)) >= (unsigned long long)deviceProp.totalGlobalMem) {
            fprintf(stderr, "Memory overload!\n");
            exit(EXIT_FAILURE);
      }
      
      
      if( threadsPerBlock >= deviceProp.maxThreadsPerBlock){
            fprintf(stderr, "Threads overload!\n");
            exit(EXIT_FAILURE);
      }

      if( blocksPerGrid >= deviceProp.maxGridSize[0]){
    	    fprintf(stderr, "Grid overload!\n");
    	    exit(EXIT_FAILURE);
      }
      
      add<<<threadsPerBlock, blocksPerGrid >>>(N, x);

      // Wait for GPU to finish before accessing on host
      hipDeviceSynchronize();

      // Free memory
      hipFree(x);

      return 0;
    }
