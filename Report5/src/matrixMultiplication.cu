#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define MIN(a, b) (a<b?a:b)
#define BLOCK_SIZE 32

struct Matrix {
	int height;
	int width;
	int *el;
	int stride;
	
	__host__ __device__
	Matrix(int height, int width, int stride ): height(height), width(width),stride(stride){}
	
	__host__ __device__	
	Matrix(const Matrix &a): height(a.height), width(a.width),el(a.el),stride(a.stride){}
	
	__device__ 
	float getElement(int row, int col){
		return el[row * stride + col];
	}
	
	__host__ __device__
	void operator =(const Matrix &a){height = a.height; width = a.width; el = a.el; stride = a.stride;}

	__device__
	 void setElement(int row, int col, int val){
		el[row * stride + col] = val;
	}
	
	__device__
	 Matrix cutMatrix(int row, int col){
		Matrix tmp(BLOCK_SIZE, BLOCK_SIZE, stride);
		
		tmp.el = &el[stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
		return tmp;
	}
	
	__host__  
	 void writeOut(){
		for(int i = 0; i < height; i++){
			std::cout<<"| ";
		 	for(int j = 0; j < width; j++){
		 		std::cout<<el[i * width + j]<<" ";
		 	}
			std::cout<<"|"<<std::endl;
		 }
		std::cout<<"\n";
	}
};


__global__
void MatrixMulKernel(Matrix a,Matrix b, Matrix c) {
	int cutRow = blockIdx.y ;
	int cutCol = blockIdx.x;

	int fRow = blockIdx.y * blockDim.y + threadIdx.y;
	int fCol = blockIdx.x * blockDim.x + threadIdx.x;	
	int row = threadIdx.y;
	int col = threadIdx.x;
	
	int temp = 0;
	
	Matrix cutMatC = c.cutMatrix(cutRow, cutCol);
	
	for( int v = 0; v < ((a.width + BLOCK_SIZE - 1)/BLOCK_SIZE); ++v){
		Matrix cutMatA = a.cutMatrix(cutRow, v);	//cut input matrix vector which can fit inside block
		Matrix cutMatB = b.cutMatrix(v, cutCol);	
	
		__shared__ int A[BLOCK_SIZE][BLOCK_SIZE];	//Matrix wchich can share memory between threads
		__shared__ int B[BLOCK_SIZE][BLOCK_SIZE];
		if((row  < a.height) && ((col + v * BLOCK_SIZE) < a.width)){ 		
			A[row][col] = cutMatA.getElement(row, col);
		}
		else{
			A[row][col] = 0;
		}

		if((col < b.width) && ((row + v * BLOCK_SIZE) < b.height)){
			B[row][col] = cutMatB.getElement(row, col);
		}
		else{
			B[row][col] = 0;
		}

		__syncthreads();				//make sure that every metrix is filled
	
		for (int i = 0; i < BLOCK_SIZE; ++i){
			temp += A[row][i] * B[i][col];
		}		
		__syncthreads();
	
	}

	if(fRow < c.height && fCol < c.width)
		c.setElement(fRow, fCol, temp);
}

int main(){
	int N = 12;
	int M = 8;
	Matrix a(N, M, M), g(M, N, N), ag(N, N, N);
	
	hipError_t err = hipSuccess;	
	
	dim3 threadsPerBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 blocksPerGrid((a.width + threadsPerBlock.x - 1)  / threadsPerBlock.x ,(g.height + threadsPerBlock.y - 1) / threadsPerBlock.y  );
	
	hipMallocManaged(&a.el,N * M * sizeof(int));
	hipMallocManaged(&g.el, N * M * sizeof(int));
	hipMallocManaged(&ag.el, N * N * sizeof(int));
	
	for(int i = 0; i < M; i++){
		for(int j = 0; j<N; j++){
			a.el[i*N+j] = 1;  
			g.el[i*N+j] = 2;
		}
	}

	MatrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>( a, g, ag);
	
     	hipDeviceSynchronize();
	if (err != hipSuccess){	
        	fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
	
	a.writeOut();	
	g.writeOut();
	ag.writeOut();

			
	hipFree(a.el);
	hipFree(g.el);
	hipFree(ag.el);
}	
