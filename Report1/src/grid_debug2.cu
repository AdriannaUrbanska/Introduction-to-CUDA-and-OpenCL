// a simple code to understand the grid and block layout
// and thread numbering scheme

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void checkIndex(void) {
  printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
  "gridDim:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
  blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
  gridDim.x,gridDim.y,gridDim.z);
}

__global__ void
vectorAdd(int *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        A[i] = A[i] * 2;
    }
}

int main(int argc, char **argv) {

  int nElem = 102173;
  int *h_vect = (int *)malloc(nElem * sizeof(int));
  int *d_vect = NULL;

  for (int i = 0; i < nElem; ++i)
     {
         h_vect[i] = rand();
     }

  hipMalloc((void **)&d_vect, nElem * sizeof(int));

  int threadsPerBlock = 8388608;
  int blocksPerGrid = (nElem+threadsPerBlock-1)/threadsPerBlock;
  // check grid and block dimension from host side
  //printf("grid.x %d grid.y %d grid.z %d\n",grid.x, grid.y, grid.z);
  //printf("block.x %d block.y %d block.z %d\n",block.x, block.y, block.z);

  hipMemcpy(d_vect, h_vect, nElem * sizeof(int), hipMemcpyHostToDevice);

  vectorAdd<<<threadsPerBlock, blocksPerGrid>>>(d_vect, nElem);
  hipDeviceSynchronize();

  hipMemcpy(h_vect, d_vect, nElem * sizeof(int), hipMemcpyDeviceToHost);


  // check grid and block dimension from device side
  //checkIndex <<<grid, block>>> ();
  // reset device before you leave
  hipDeviceReset();
  free(h_vect);
  hipFree(d_vect);
  return(0);

}

