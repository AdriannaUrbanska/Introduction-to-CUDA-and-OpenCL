#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <vector>
#define N 511
#define BLOCK_SIZE 16

__global__ void  reduceSum(int *ada, int *gabrys){
	__shared__ int partialSum[2 * BLOCK_SIZE];
	unsigned int t = threadIdx.x;
	unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

	if(start + t < N){
		partialSum[t] = ada[start + t];
	}
	else{
		partialSum[t] = 0;
	}

	if (start + BLOCK_SIZE + t < N){
		partialSum[BLOCK_SIZE + t] = ada[start + BLOCK_SIZE + t];
	}
	else{
		partialSum[BLOCK_SIZE + t] = 0;
	}   

	for(unsigned int stride = 1; stride <= BLOCK_SIZE ; stride *= 2){
		__syncthreads();
		if (t % stride == 0 ){
			partialSum[2*t] += partialSum[2*t + stride];
		}	
	} 

	if(t == 0){
		gabrys[blockIdx.x] = partialSum[0];
	}

	__syncthreads();
}

int main(void){
	int * ada, * gabrys;
	hipMallocManaged(&ada, N * sizeof(int));
	hipMallocManaged(&gabrys, N * sizeof(int));

	for(int i = 0; i < N; i++){
		ada[i] = 1;
	}
	dim3 threadsPerBlock(BLOCK_SIZE);
	dim3 blocksPerGrid((N + BLOCK_SIZE - 1)/BLOCK_SIZE);
		
	reduceSum<<<blocksPerGrid, threadsPerBlock>>>(ada,gabrys);
	hipDeviceSynchronize();
	
	int count = 1;

	for(int i = 0; i<count; i++){
		reduceSum<<<blocksPerGrid, BLOCK_SIZE>>>(gabrys,gabrys);
		hipDeviceSynchronize();
	}
	
	for(int i = 0; i<10; i++)
	std::cout<<gabrys[i]<<std::endl;
}
