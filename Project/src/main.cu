#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : CudaProject.cu
 Author      : Adrianna Urbańska, Gabriel Chęć
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32

class CudaObject {

	int threadsPerBlock = 256;
	int blocksPerGrid = 32;
public:
	int size_x;
	int size_y;
	int *data;
	int stride;
	int bytes;
	int SM = 1;
	
	__host__ __device__
	CudaObject(int x, int y, int stride ): size_x(x), size_y(y),stride(stride){}
	
	__host__ __device__	
	CudaObject(const CudaObject &a): size_x(a.size_x), size_y(a.size_y),data(a.data),stride(a.stride){}
	
	__device__ 
	int getElement(int row, int col){
		return data[row * stride + col];
	}
	
	__host__ __device__
	void operator =(const CudaObject &a){size_x = a.size_x; size_y = a.size_y; data = a.data; stride = a.stride;}

	__device__
	 void setElement(int row, int col, int val){
		data[row * stride + col] = val;
	}
	
	__device__
	CudaObject cutMatrix(int row, int col){
		CudaObject tmp(BLOCK_SIZE, BLOCK_SIZE, stride);
		
		tmp.data = &data[stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
		return tmp;
	}
	
	__host__  
	 void writeOut(){
		for(int i = 0; i < size_x; i++){
			std::cout<<"| ";
		 	for(int j = 0; j < size_y; j++){
		 		std::cout<<data[i * size_y + j]<<" ";
		 	}
			std::cout<<"|"<<std::endl;
		 }
		std::cout<<"\n";
	}

	void setData(int *data, int x, int y){
		this->size_x = x;
		this->size_y = y;
		this->bytes = x * y * sizeof(int);
		hipMallocManaged(&this->data, this->bytes);
		memcpy(this->data, data, this->bytes);
	}

	void setSize(int x, int y){
		this->size_x = x;
		this->size_y = y;
		this->bytes = x * y * sizeof(int);
		hipMallocManaged(&this->data, this->bytes);
	}


	void addCpu(CudaObject &fData, CudaObject &sData){
		if(fData.size_x == sData.size_x && fData.size_y == sData.size_y){
			hipMemPrefetchAsync(this->data, this->bytes, hipCpuDeviceId);
			hipMemPrefetchAsync(fData.data, fData.bytes, hipCpuDeviceId);
			hipMemPrefetchAsync(sData.data, sData.bytes, hipCpuDeviceId);

			this->setSize(sData.size_x, sData.size_y);

			for(int i = 0; i < sData.size_x; i++){
				for(int j = 0; j < sData.size_y; j++){
					this->data[i * this->size_x + j] = fData.data[i * this->size_x + j] + sData.data[i * this->size_x + j];
				}
			}
		}
	}

	void subCpu(CudaObject &fData, CudaObject &sData){
		if(fData.size_x == sData.size_x && fData.size_y == sData.size_y){
			hipMemPrefetchAsync(this->data, this->bytes, hipCpuDeviceId);
			hipMemPrefetchAsync(fData.data, fData.bytes, hipCpuDeviceId);
			hipMemPrefetchAsync(sData.data, sData.bytes, hipCpuDeviceId);

			this->setSize(sData.size_x, sData.size_y);

			for(int i = 0; i < sData.size_x; i++){
				for(int j = 0; j < sData.size_y; j++){
					this->data[i * this->size_x + j] = fData.data[i * this->size_x + j] - sData.data[i * this->size_x + j];
				}
			}
		}
	}

	void mulCpu(CudaObject &fData, CudaObject &sData){
		int y_s = sData.size_y;
		int y_f = fData.size_y;
  		for(int i = 0; i < size_x; i++ ){
    		for(int j = 0; j < size_y; j++ ){
      			int s = 0;
      			for(int k = 0; k < y_f; k++ ) 
					s += fData.data[i * y_f + k] * sData.data[k * y_s + j];
				this->data[i * y_s + j] = s;
    		}
		}
	}

};

__global__ void add(int *fData, int *sData, int *oData, int x, int y){

		  int index = threadIdx.x + blockIdx.x * blockDim.x;
		  int stride = blockDim.x * gridDim.x;

		  for(int i = index; i < x*y; i += stride)
		  {
			oData[i] = fData[i] + sData[i];
		  }
}

__global__ void sub(int *fData, int *sData, int *oData, int x, int y){

	  int index = threadIdx.x + blockIdx.x * blockDim.x;
	  int stride = blockDim.x * gridDim.x;

	  for(int i = index; i < x*y; i += stride)
	  {
	    oData[i] = fData[i] - sData[i];
	  }
}	

__global__
void MatrixMulKernel(CudaObject a,CudaObject b, CudaObject c) {
	int cutRow = blockIdx.y ;
	int cutCol = blockIdx.x;

	int fRow = blockIdx.y * blockDim.y + threadIdx.y;
	int fCol = blockIdx.x * blockDim.x + threadIdx.x;	
	int row = threadIdx.y;
	int col = threadIdx.x;
	
	int temp = 0;
	
	CudaObject cutMatC = c.cutMatrix(cutRow, cutCol);
	
	for( int v = 0; v < ((a.size_y + BLOCK_SIZE - 1)/BLOCK_SIZE); ++v){
		CudaObject cutMatA = a.cutMatrix(cutRow, v);	//cut input matrix vector which can fit inside block
		CudaObject cutMatB = b.cutMatrix(v, cutCol);	
	
		__shared__ int A[BLOCK_SIZE][BLOCK_SIZE];	//Matrix wchich can share memory between threads
		__shared__ int B[BLOCK_SIZE][BLOCK_SIZE];
		if((row  < a.size_x) && ((col + v * BLOCK_SIZE) < a.size_y)){ 		
			A[row][col] = cutMatA.getElement(row, col);
		}
		else{
			A[row][col] = 0;
		}

		if((col < b.size_y) && ((row + v * BLOCK_SIZE) < b.size_x)){
			B[row][col] = cutMatB.getElement(row, col);
		}
		else{
			B[row][col] = 0;
		}

		__syncthreads();				//make sure that every metrix is filled
	
		for (int i = 0; i < BLOCK_SIZE; ++i){
			temp += A[row][i] * B[i][col];
		}		
		__syncthreads();
	
	}

	if(fRow < c.size_x && fCol < c.size_y)
		c.setElement(fRow, fCol, temp);
}

void OperationsInfo()
{
	std::cout<<"Choose an operation:"<<std::endl;
	std::cout<<"1. Matrix addition on CPU"<<std::endl;
	std::cout<<"2. Matrix addition on GPU"<<std::endl;
	std::cout<<"3. Matrix substraction on CPU"<<std::endl;
	std::cout<<"4. Matrix substraction on GPU"<<std::endl;
	std::cout<<"5. Matrix multiplication on CPU"<<std::endl;
	std::cout<<"6. Matrix multiplication on GPU"<<std::endl;
}

void Init(CudaObject &oData, int val)
{
	int x = oData.size_x;
	int y = oData.size_y;

	for(int i = 0; i < y; i++){
		for(int j = 0; j<x; j++){
			oData.data[i*x+j] = val;  
		}
	}
}

int main(){

	int operation;
	int N_1, N_2, M_1, M_2;
	int val_1, val_2;

	std::cout<<"Enter the values of size_x, size_y of the first matrix and value to filled matrix:"<<std::endl;
	std::cin>>N_1;
	std::cin>>M_1;
	std::cin>>val_1;

	std::cout<<"Enter the values of size_x, size_y of the second matrix and value to filled matrix::"<<std::endl;
	std::cin>>N_2;
	std::cin>>M_2;
	std::cin>>val_2;

	CudaObject fData(N_1, M_1, M_1), sData(N_2, M_2, M_2), oData(N_1, M_2, M_2);

	hipMallocManaged(&fData.data,N_1 * M_1 * sizeof(int));
	hipMallocManaged(&sData.data, N_2 * M_2 * sizeof(int));
	hipMallocManaged(&oData.data, N_1 * M_2 * sizeof(int));

	Init(fData,val_1);
	Init(sData,val_2);

	fData.writeOut();	
	sData.writeOut();

	OperationsInfo();
	std::cin>>operation;

	dim3 threadsPerBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 blocksPerGrid((fData.size_y + threadsPerBlock.x - 1)/threadsPerBlock.x, (sData.size_x + threadsPerBlock.y - 1)/threadsPerBlock.y);

	switch(operation)
	{
		case 1:
			if(fData.size_x != sData.size_x || fData.size_y != sData.size_y){
				std::cout<<"Matrices sizes have to be equal!"<<std::endl;
			}
			else{
				oData.addCpu(fData,sData);
				oData.writeOut();
			}
			break;
		case 2:
			if(fData.size_x != sData.size_x || fData.size_y != sData.size_y){
				std::cout<<"Matrices sizes have to be equal!"<<std::endl;
			}
			else{
				add<<<blocksPerGrid,threadsPerBlock>>>(fData.data, sData.data, oData.data, oData.size_x, oData.size_y);
				hipDeviceSynchronize();
				oData.writeOut();
			}
			break;
		case 3:
			if(fData.size_x != sData.size_x || fData.size_y != sData.size_y){
				std::cout<<"Matrices sizes have to be equal!"<<std::endl;
			}
			else{
				oData.subCpu(fData,sData);
				oData.writeOut();
			}
			break;
		case 4:
			if(fData.size_x != sData.size_x || fData.size_y != sData.size_y){
				std::cout<<"Matrices sizes have to be equal!"<<std::endl;
			}
			else{
				sub<<<blocksPerGrid,threadsPerBlock>>>(fData.data, sData.data, oData.data, oData.size_x, oData.size_y);
				hipDeviceSynchronize();
				oData.writeOut();
			}
			break;
		case 5:
			if(fData.size_y != sData.size_x){
				std::cout<<"Size_x of the first matrix and size_y of the second matrix have to be equal!"<<std::endl;
			}
			else{
				oData.mulCpu(fData,sData);
				oData.writeOut();
			}
			break;
		case 6:
			if(fData.size_y != sData.size_x){
				std::cout<<"Size_x of the first matrix and size_y of the second matrix have to be equal!"<<std::endl;
			}
			else{
				MatrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(fData, sData, oData);	
				hipDeviceSynchronize();
				oData.writeOut();
			}			
			break;

		default:
			std::cout<<"Wrong number entered!"<<std::endl;
			break;
	}
	
	hipError_t err = hipSuccess;	
		
	if (err != hipSuccess){	
        	fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
			
	hipFree(fData.data);
	hipFree(sData.data);
	hipFree(oData.data);
}	
