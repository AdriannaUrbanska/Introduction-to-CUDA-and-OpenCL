/*
 ============================================================================
 Name        : CudaProject.cu
 Author      : Adrianna Urbańska, Gabriel Chęć
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <stdio.h>
#include <math.h>
#include "CudaObject.h"
#include <stdlib.h>

int main(void)
{
	int * a, *b;
	int size = 2<<14;
	size_t s = size * size * sizeof(int);

	a = (int *)malloc(s);
	b = (int *)malloc(s);

	//hipMallocManaged(&a, s);
	//hipMallocManaged(&b, s);

	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			a[i * size + j] = 2;
			b[i * size + j] = 3;
		}
	}

	CudaObject a_mat(a, size, size);
	CudaObject b_mat(b, size, size);
	CudaObject c_mat;

	free(b);
	free(a);
	hipFree(a);
	hipFree(b);
	c_mat.addGpu(a_mat,b_mat);
	//c_mat.show();
	return 0;
}
