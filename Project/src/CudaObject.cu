#include "hip/hip_runtime.h"
#include "CudaObject.h"
#include <iostream>
#include <stdio.h>


CudaObject::CudaObject( int * data, int x, int y){
	this->size_x = x;
	this->size_y = y;
	this->bytes = x * y * sizeof(int);
	hipMallocManaged(&this->data, this->bytes);
	memcpy(this->data, data, this->bytes);
	hipGetDevice(&this->deviceId);
	hipDeviceGetAttribute(&this->SM, hipDeviceAttributeMultiprocessorCount,this->deviceId);
	hipGetDevice(&this->deviceId);
	this->threadsPerBlock = 256;
	this->blocksPerGrid = 32 * this->SM;
}

CudaObject::CudaObject(){
	this->size_x = 0;
	this->size_y = 0;
	this->bytes = 0;
	this->data = 0;

	hipGetDevice(&this->deviceId);
	hipDeviceGetAttribute(&this->SM, hipDeviceAttributeMultiprocessorCount,this->deviceId);
	this->threadsPerBlock = 256;
		this->blocksPerGrid = 32 * this->SM;

}

CudaObject::~CudaObject() { hipFree(this->data);}

void CudaObject::setData(int *data, int x, int y){
		this->size_x = x;
		this->size_y = y;
		this->bytes = x * y * sizeof(int);
		hipMallocManaged(&this->data, this->bytes);
		memcpy(this->data, data, this->bytes);
}

void CudaObject::setSize(int x, int y){
		this->size_x = x;
		this->size_y = y;
		this->bytes = x * y * sizeof(int);
		hipMallocManaged(&this->data, this->bytes);
}

void CudaObject::addGpu(CudaObject &fData, CudaObject &sData){
	if(fData.size_x == sData.size_x && fData.size_y == sData.size_y){
		this->setSize(sData.size_x, sData.size_y);

		hipMemPrefetchAsync(this->data, this->bytes, this->deviceId);
		hipMemPrefetchAsync(sData.data, sData.bytes, this->deviceId);
		hipMemPrefetchAsync(fData.data, fData.bytes, this->deviceId);
		add<<<this->blocksPerGrid, this->threadsPerBlock>>>(fData.data, sData.data, this->data,sData.size_x, sData.size_y);

		hipDeviceSynchronize();
	}
}


void CudaObject::addCpu( CudaObject &fData, CudaObject &sData){
	if(fData.size_x == sData.size_x && fData.size_y == sData.size_y){
		hipMemPrefetchAsync(this->data, this->bytes, hipCpuDeviceId);
		hipMemPrefetchAsync(fData.data, fData.bytes, hipCpuDeviceId);
		hipMemPrefetchAsync(sData.data, sData.bytes, hipCpuDeviceId);

		this->setSize(sData.size_x, sData.size_y);

		for(int i = 0; i < sData.size_x; i++){
			for(int j = 0; j < sData.size_y; j++){
				this->data[i * this->size_x + j] = fData.data[i * this->size_x + j] + sData.data[i * this->size_x + j];
			}
		}
	}
}

void CudaObject::subGpu(CudaObject &fData, CudaObject &sData){
	if(fData.size_x == sData.size_x && fData.size_y == sData.size_y){
		this->setSize(sData.size_x, sData.size_y);

		hipMemPrefetchAsync(this->data, this->bytes, this->deviceId);
		hipMemPrefetchAsync(sData.data, sData.bytes, this->deviceId);
		hipMemPrefetchAsync(fData.data, fData.bytes, this->deviceId);
		sub<<<this->blocksPerGrid, this->threadsPerBlock>>>(fData.data, sData.data, this->data,sData.size_x, sData.size_y);

		hipDeviceSynchronize();
	}
}


void CudaObject::subCpu( CudaObject &fData, CudaObject &sData){
	if(fData.size_x == sData.size_x && fData.size_y == sData.size_y){
		hipMemPrefetchAsync(this->data, this->bytes, hipCpuDeviceId);
		hipMemPrefetchAsync(fData.data, fData.bytes, hipCpuDeviceId);
		hipMemPrefetchAsync(sData.data, sData.bytes, hipCpuDeviceId);

		this->setSize(sData.size_x, sData.size_y);

		for(int i = 0; i < sData.size_x; i++){
			for(int j = 0; j < sData.size_y; j++){
				this->data[i * this->size_x + j] = fData.data[i * this->size_x + j] - sData.data[i * this->size_x + j];
			}
		}
	}
}

void CudaObject::show(){
	hipMemPrefetchAsync(this->data, this->bytes, hipCpuDeviceId);

	for(int i = 0; i < this->size_x; i++){
		std::cout<<"|\t";
		for(int j = 0; j < this->size_y; j++){
			std::cout<<data[i * this->size_x + j]<< "\t";
		}
		std::cout<<" |"<<std::endl;
	}
}

__global__ void add(int *fData, int *sData, int *oData, int x, int y){

	  int index = threadIdx.x + blockIdx.x * blockDim.x;
	  int stride = blockDim.x * gridDim.x;

	  for(int i = index; i < x*y; i += stride)
	  {
	    oData[i] = fData[i] + sData[i];
	  }
}

__global__ void sub(int *fData, int *sData, int *oData, int x, int y){

	  int index = threadIdx.x + blockIdx.x * blockDim.x;
	  int stride = blockDim.x * gridDim.x;

	  for(int i = index; i < x*y; i += stride)
	  {
	    oData[i] = fData[i] - sData[i];
	  }
}

