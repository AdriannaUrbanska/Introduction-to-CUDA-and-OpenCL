#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define MIN(a, b) (a < b ? a : b)

__global__
void MatrixMulKernel(const int* M,const int* N, int* P, int Width) {
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	int Col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((Row < Width) && (Col < Width))
	{
		int Pvalue = 0; 
		for (int k = 0; k < Width; ++k)
		{
			Pvalue += M[Row * Width + k] * N[k * Width + Col];
		}
		P[Row * Width + Col] = Pvalue;
	}

}

int main(){
	int *a, *g, *ag;
	int N = 1000;
	int thread = 256;
	dim3 threadsPerBlock(thread,thread);
	dim3 blocksPerGrid(MIN(32, (N + thread - 1) / thread), MIN(32, (N + thread - 1) / thread));

	hipMallocManaged(&a,N * N * sizeof(int));
	hipMallocManaged(&g, N * N * sizeof(int));
	hipMallocManaged(&ag, N * N * sizeof(int));

	for(int i = 0; i < N; i++){
		for(int j = 0; j<N; j++){
			a[i*N+j] = 2; //było a[i*j+i] zamiast a[i*size+j]	aaaaaaaaaaaaaaaa kurde głupi błąd 
			g[i*N+j] = 3;
		}
	 }
	 MatrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>( a, g, ag, N);
     hipDeviceSynchronize();


//-----------------------------------------------------Writing out-------------------------------------------------------
	 for(int i = 0; i < N; i++){
		std:: cout<<"| ";
	 		for(int j = 0; j < N; j++){
	 			std::cout<<a[i * size + j]<<" ";
	 		}
			std:: cout<<"|"<<std::endl;
	 	}
	 std::cout<<"--------------------------------"<<std::endl;
	 for(int i = 0; i < N; i++){
		 std::cout<<"| ";
	 		for(int j = 0; j < N; j++){
	 			std::cout<<g[i * size + j]<<" ";
	 		}
			 std::cout<<"|"<<std::endl;
	 	}
	 std::cout<<"--------------------------------"<<std::endl;
	 for(int i = 0; i < N; i++){
		 std::cout<<"| ";
	 		for(int j = 0; j < N; j++){
	 			std::cout<<ag[i * size + j]<<" ";
	 		}
			 std::cout<<"|"<<std::endl;
	 	}
	    hipFree(a);
	    hipFree(g);
	    hipFree(ag);
}	
